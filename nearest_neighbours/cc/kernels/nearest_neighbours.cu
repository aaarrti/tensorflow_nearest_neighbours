#include "hip/hip_runtime.h"


#include "nearest_neighbours.h"
#include "tensorflow/core/util/gpu_kernel_helper.h"
#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"

#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU
#endif


namespace tensorflow {
  namespace functor {

    typedef Eigen::GpuDevice GPUDevice;

    // Define the CUDA kernel.
    __global__ void NearestNeighboursCudaKernel(
        const int32_t batch_size,
        const int32_t vocab_size,
        const int32_t sequence_length,
        const int32_t embedding_dim,
        const tensorflow::Tensor *token_embeddings,
        const tensorflow::Tensor *embedding_matrix_shaped,
        const Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> eigen_embedding_matrix,
        tensorflow::Tensor *output_tensor) {

      const auto sequence = Eigen::Map<const Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>>(
          token_embeddings->SubSlice(threadId.x).flat<float>().data(), vocab_size, embedding_dim);
      auto distances = std::vector<float>(vocab_size);
      const auto embedding = sequence.row(threadId.y);
      // Find index of the smallest distance
      auto argmin = nearest_neighbour_index(vocab_size, embedding, eigen_embedding_matrix);
      // Fill the output
      for (auto i = 0; i != embedding_dim; i++) {
        output_shaped({threadId.x, threadId.y, i}) = embedding_matrix_shaped({argmin, i});
      }
    }


    template<>
    struct NearestNeighboursFunctor<GPUDevice> {
      void operator()(const GPUDevice &device,
                      const tensorflow::Tensor *token_embeddings,
                      const tensorflow::Tensor *embedding_matrix,
                      tensorflow::Tensor *output_tensor) {

        const auto batch_size = static_cast<int32_t>(token_embeddings->dim_size(0));
        const auto vocab_size = static_cast<int32_t>(embedding_matrix->dim_size(0));
        const auto sequence_length = static_cast<int32_t>(token_embeddings->dim_size(1));
        const auto embedding_dim = static_cast<int32_t>(token_embeddings->dim_size(2));

        const auto block_count = batch_size;
        const auto thread_per_block = sequence_length;

        auto embedding_matrix_shaped = embedding_matrix->shaped<float, 2>({vocab_size, embedding_dim});
        const auto eigen_embedding_matrix = Eigen::Map<const Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> >(
            embedding_matrix->flat<float>().data(), vocab_size, embedding_dim);

        NearestNeighboursCudaKernel<<<block_count, thread_per_block, 0, device.stream()>>>(
            batch_size, vocab_size, sequence_length, embedding_dim,
            token_embeddings, embedding_matrix_shaped, eigen_embedding_matrix);
      }
    };

// Explicitly instantiate functors for the types of OpKernels registered.
    template
    struct NearestNeighboursFunctor<GPUDevice>;
  } // end namespace functor
} // end namespace tensorflow